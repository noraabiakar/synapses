#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace Ampa_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
\
auto _pp_var_u0 __attribute__((unused)) = params_.globals[0];\
\
auto* _pp_var_C __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_O __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_D __attribute__((unused)) = params_.state_vars[2];\
auto* _pp_var_delay __attribute__((unused)) = params_.state_vars[3];\
auto* _pp_var_v __attribute__((unused)) = params_.state_vars[4];\
auto* _pp_var_g __attribute__((unused)) = params_.state_vars[5];\
auto* _pp_var_T __attribute__((unused)) = params_.state_vars[6];\
auto* _pp_var_Trelease __attribute__((unused)) = params_.state_vars[7];\
auto* _pp_var_x __attribute__((unused)) = params_.state_vars[8];\
auto* _pp_var_tsyn __attribute__((unused)) = params_.state_vars[9];\
auto* _pp_var_Mres __attribute__((unused)) = params_.state_vars[10];\
auto* _pp_var_NTdiffusion __attribute__((unused)) = params_.state_vars[11];\
auto* _pp_var_numpulses __attribute__((unused)) = params_.state_vars[12];\
auto* _pp_var_on __attribute__((unused)) = params_.state_vars[13];\
auto* _pp_var_t0 __attribute__((unused)) = params_.state_vars[14];\
auto* _pp_var_y __attribute__((unused)) = params_.state_vars[15];\
auto* _pp_var_z __attribute__((unused)) = params_.state_vars[16];\
auto* _pp_var_u __attribute__((unused)) = params_.state_vars[17];\
\
auto* _pp_var_gmax __attribute__((unused)) = params_.parameters[0];\
auto* _pp_var_Cdur __attribute__((unused)) = params_.parameters[1];\
auto* _pp_var_Erev __attribute__((unused)) = params_.parameters[2];\
auto* _pp_var_kB __attribute__((unused)) = params_.parameters[3];\
auto* _pp_var_r1FIX __attribute__((unused)) = params_.parameters[4];\
auto* _pp_var_r6FIX __attribute__((unused)) = params_.parameters[5];\
auto* _pp_var_r2 __attribute__((unused)) = params_.parameters[6];\
auto* _pp_var_r5 __attribute__((unused)) = params_.parameters[7];\
auto* _pp_var_tau_1 __attribute__((unused)) = params_.parameters[8];\
auto* _pp_var_tau_rec __attribute__((unused)) = params_.parameters[9];\
auto* _pp_var_tau_facil __attribute__((unused)) = params_.parameters[10];\
auto* _pp_var_U __attribute__((unused)) = params_.parameters[11];\
auto* _pp_var_Tmax __attribute__((unused)) = params_.parameters[12];\
auto* _pp_var_M __attribute__((unused)) = params_.parameters[13];\
auto* _pp_var_R __attribute__((unused)) = params_.parameters[14];\
auto* _pp_var_Diff __attribute__((unused)) = params_.parameters[15];\
auto* _pp_var_lamd __attribute__((unused)) = params_.parameters[16];\
\
arb_value_type* __attribute__((unused)) _pp_var_tspike[50] = {\
    pp->state_vars[18], pp->state_vars[19], pp->state_vars[20], pp->state_vars[21], pp->state_vars[22],\
    pp->state_vars[23], pp->state_vars[24], pp->state_vars[25], pp->state_vars[26], pp->state_vars[27],\
    pp->state_vars[28], pp->state_vars[29], pp->state_vars[30], pp->state_vars[31], pp->state_vars[32],\
    pp->state_vars[33], pp->state_vars[34], pp->state_vars[35], pp->state_vars[36], pp->state_vars[37],\
    pp->state_vars[38], pp->state_vars[39], pp->state_vars[40], pp->state_vars[41], pp->state_vars[42],\
    pp->state_vars[43], pp->state_vars[44], pp->state_vars[45], pp->state_vars[46], pp->state_vars[47],\
    pp->state_vars[48], pp->state_vars[49], pp->state_vars[50], pp->state_vars[51], pp->state_vars[52],\
    pp->state_vars[53], pp->state_vars[54], pp->state_vars[55], pp->state_vars[56], pp->state_vars[57],\
    pp->state_vars[58], pp->state_vars[59], pp->state_vars[60], pp->state_vars[61], pp->state_vars[62],\
    pp->state_vars[63], pp->state_vars[64], pp->state_vars[65], pp->state_vars[66], pp->state_vars[67],\
};\
\
arb_value_type* ___attribute__((unused)) pp_var_PRE[50] = {\
    pp->state_vars[68], pp->state_vars[69], pp->state_vars[70], pp->state_vars[71], pp->state_vars[72],\
    pp->state_vars[73], pp->state_vars[74], pp->state_vars[75], pp->state_vars[76], pp->state_vars[77],\
    pp->state_vars[78], pp->state_vars[79], pp->state_vars[80], pp->state_vars[81], pp->state_vars[82],\
    pp->state_vars[83], pp->state_vars[84], pp->state_vars[85], pp->state_vars[86], pp->state_vars[87],\
    pp->state_vars[88], pp->state_vars[89], pp->state_vars[90], pp->state_vars[91], pp->state_vars[92],\
    pp->state_vars[93], pp->state_vars[94], pp->state_vars[95], pp->state_vars[96], pp->state_vars[97],\
    pp->state_vars[98], pp->state_vars[99], pp->state_vars[100], pp->state_vars[101], pp->state_vars[102],\
    pp->state_vars[103], pp->state_vars[104], pp->state_vars[105], pp->state_vars[106], pp->state_vars[107],\
    pp->state_vars[108], pp->state_vars[109], pp->state_vars[110], pp->state_vars[111], pp->state_vars[112],\
    pp->state_vars[113], pp->state_vars[114], pp->state_vars[115], pp->state_vars[116], pp->state_vars[117],\
};\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        auto vec_dii_     = _pp_var_vec_di[node_indexi_];
        arb_value_type t = _pp_var_vec_t[vec_dii_];
        _pp_var_C[tid_] =  1.0;
        _pp_var_O[tid_] =  0.;
        _pp_var_D[tid_] =  0.;
        _pp_var_delay[tid_] =  0.;
        _pp_var_T[tid_] =  0.;
        _pp_var_Trelease[tid_] =  0.;
        _pp_var_Mres[tid_] =  1.6605778811026237e-06*_pp_var_M[tid_];
        _pp_var_numpulses[tid_] =  0.;
        _pp_var_on[tid_] =  0.;
        _pp_var_y[tid_] =  0.;
        _pp_var_z[tid_] =  0.;
        _pp_var_u[tid_] = _pp_var_u0;
        _pp_var_tsyn[tid_] = t;
        _pp_var_tspike[0][tid_] =  1e12; // EDITTED
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];

        // Read 
        double O  = _pp_var_O[tid_];
        double D  = _pp_var_D[tid_];
        double r2 = _pp_var_r2[tid_];
        double r5 = _pp_var_r5[tid_];

        double tr = _pp_var_Trelease[i_];
        double k  = _pp_var_kB[i_];
        double ratio = std::pow(tr,2)/std::pow((tr+k),2);

        double r1 = _pp_var_r1FIX[i_] * ratio;
        double r6 = _pp_var_r6FIX[i_] * ratio;

        // Solve ODEs 
        double t0  =  -r6 * dt;
        double t1  =  -r1 * dt;
        double t2  =  1.0 + r5*dt;
        double t3  =  1.0 + r2*dt;
        double t4  = t3 - t1;
        double t5  = t3 - O;
        double t6  = (t2 * t4) - (t3 * t0);
        double t7  = (t2 * t5) - (t3 * D);
        double t8  = t6 * t2;
        double t9  = (t6 * D) - (t0 * t7);
        double t10 = t6 * t3;


        // Update 
        if (tr > 0) {
            _pp_var_C[tid_] = t7 / t6;
            _pp_var_D[tid_] = t9 / t8;
            _pp_var_O[tid_] = t11 / t10;
        }
        _pp_var_delay[tid_] -= dt;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned lane_mask_ = arb::gpu::ballot(0xffffffff, tid_<n_);
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        auto vec_dii_ = _pp_var_vec_di[node_indexi_];

        arb_value_type t = _pp_var_vec_t[vec_dii_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];

        // START EDIT
        // Read
        const auto mres = _pp_var_Mres[tid_];
        const auto r = _pp_var_R[tid_];
        const auto diff = _pp_var_Diff[tid_];
        const auto lamd = _pp_var_lamd[tid_];
        const auto numpulses = (int)_pp_var_numpulses[tid_];

        // Calculate
        const auto rsq = r*r;  
        const auto diff_4 = diff*4;
        const auto lamd_scaled = (1e-3)*lamd;

        auto NTdiffWave = _pp_var_T[tid_];
        const auto max_pulses = std::min(numpulses, 50); 
	for (unsigned pulse = 0; pulse < max_pulses; ++pulse) {
            auto ts     = _pp_var_tspike[pulse][tid_]; 

            auto delta_t = t - ts; 
            if (delta_t > 0.) {
                auto pre = _pp_var_PRE[pulse][tid_]; 
                auto invariant = delta_t*diff_4; 
                NTdiffWave += pre*mres*std::exp(-rsq/invariant)/(3.14159*invariant*lamd_scaled);
            }
        }

        // Update
        _pp_var_Trelease[tid_] = NTdiffWave;
        // END EDIT

        // Reset 
        if (_pp_var_delay[tid_]< 0.) {
            _pp_var_T[tid_] =  0.;
            _pp_var_on[tid_] =  0.;
        }

        // Update
        const auto gmax = _pp_var_gmax[tid_];
        const auto O    = _pp_var_O[tid_];
        const auto Erev = _pp_var_Erev[tid_];
        const auto weight = _pp_var_weight[tid_];

        auto conductivity_ =  1e-06*gmax*O;
        auto current       =  conductivity_*(v-Erev);
        ::arb::gpu::reduce_by_key(_pp_var_weight[tid_]*conductivity_,_pp_var_vec_g, node_indexi_, lane_mask_);
        ::arb::gpu::reduce_by_key(_pp_var_weight[tid_]*current_,_pp_var_vec_i, node_indexi_, lane_mask_);
    }
}

__global__
void apply_events(arb_mechanism_ppack params_, arb_deliverable_event_stream stream) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    if(tid_<stream.n_streams) {
        auto begin = stream.events + stream.begin[tid_];
        auto end   = stream.events + stream.end[tid_];
        for (auto p = begin; p<end; ++p) {
            if (p->mech_id==_pp_var_mechanism_id) {
                auto tid_ = p->mech_index;
                auto weight = p->weight;
		auto node_indexi_ = _pp_var_node_index[tid_];
                auto vec_dii_     = _pp_var_vec_di[node_indexi_];

                arb_value_type t = _pp_var_vec_t[vec_dii_];
                if (!_pp_var_on[tid_]) {

                    // Read
                    auto z = _pp_var_z[tid_];
                    auto y = _pp_var_y[tii_];
                    auto x = _pp_var_x[tii_];
                    auto u = _pp_var_u[tii_];

                    const auto tsyn      = _pp_var_tsyn[tid_];
                    const auto tau_rec   = _pp_var_tau_rec[tid_];
                    const auto tau_1     = _pp_var_tau_1[tid_];
                    const auto tau_facil = _pp_var_tau_facil[tid_];
                    const auto U         = _pp_var_U[tid_];
                    const auto Tmax      = _pp_var_Tmax[tid_];
                    const auto numpulses = (int)_pp_var_numpulses[tid_];

                    // Modify
                    z = z*exp(-(t-tsyn)/tau_rec);
                    z = z+y*(exp(-(t-tsyn)/tau_1) - exp(-(t-tsyn)/tau_rec)) / (tau_1/tau_rec - 1.0);
                    y = y*exp(-(t-tsyn)/tau_1);
                    x =  1.0-y-z;

                    if (tau_facil> 0.) {
                        u = u*exp(-(t-tsyn)/tau_facil);
                        u = u + U*(1.0-u);
                    }
                    else {
                        u = U;
                    }
                    y = y + x * u;

                    // Update
                    _pp_var_T[tid_] = Tmax*y;
                    _pp_var_z[tid_] = z;
                    _pp_var_y[tii_] = y;
                    _pp_var_x[tii_] = x;
                    _pp_var_u[tii_] = u;

                    // START EDIT
                    auto pulse = (numpulses%50); // rolling window update
                    _pp_var_tspike[pulse][tid_] = t; 
                    _pp_var_PRE[pulse][tid_]    = y; 

                    // END EDIT

                    _pp_var_numpulses[tid_] = numpulses + 1.;
                    _pp_var_tsyn[tid_] = t;
                }
                _pp_var_delay[tid_] = _pp_var_Cdur[tid_];
            }
        }
    }
}
} // namespace

void mechanism_Ampa_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 4}, block_dim>>>(*p);
}

void mechanism_Ampa_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Ampa_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_Ampa_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_Ampa_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_Ampa_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* stream_ptr) {
    auto n = stream_ptr->n_streams;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    apply_events<<<grid_dim, block_dim>>>(*p, *stream_ptr);
}

} // namespace Ampa_catalogue
} // namespace arb
